
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add(int *a, int *b, int *c)
{
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

int main()
{
    int n;
    std::cout << "Podaj rozmiar: ";
    std::cin >> n;
    int* a_h = (int*) malloc(n * sizeof(int));
    int* b_h = (int*) malloc(n * sizeof(int));
    int* c_h = (int*) malloc(n * sizeof(int));

    for(size_t i = 0; i < n; i++)
    {
        a_h[i] = i;
        b_h[i] = 10;
    }

    for(size_t i = 0; i < n; i++)
    {
        c_h[i] = a_h[i] + b_h[i];
    }

    int *a_d = 0, *b_d, *c_d;
    //size_t size = n * sizeof(int);
    
    hipError_t rc = hipMalloc(&a_d, n * sizeof(int));
    hipMalloc(&b_d, n * sizeof(int));
    hipMalloc(&c_d, n * sizeof(int));

    hipMemcpy(a_d, a_h, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, n * sizeof(int), hipMemcpyHostToDevice);

    add<<<n, 1>>>(a_d, b_d, c_d);

    int* result = (int*) malloc(n * sizeof(int));
    hipMemcpy(result, c_d, n * sizeof(int), hipMemcpyDeviceToHost);

    if (rc != hipSuccess)
        std::cout << hipGetErrorString(rc) << std::endl;

    bool ok = true;
    for (size_t i = 0; i < n; i++)
    {
        //std::cout << c_h[i] << " " << result[i] << std::endl;
    		if(c_h[i] != result[i])
		{
			ok = false;
			break;
		}
	}
    
    std::cout << (ok ? "OK" : "ERROR") << std::endl;

    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);

    delete[] a_h;
    delete[] b_h;
    delete[] c_h;

    return 0;
}
