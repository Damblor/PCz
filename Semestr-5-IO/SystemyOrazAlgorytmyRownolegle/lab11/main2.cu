#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>

__global__ void calkaRowKernel(double* calk_d, int n)
{
    double xp, xk, h;
    xp = -1;
    xk = 1;

    h = (xk - xp) / (double)n;
    calk_d[blockIdx.x] = std::sqrt(1 - ((xp + blockIdx.x * h) *(xp + blockIdx.x * >
}


double f(double x)
{
    return std::sqrt(1 - (x * x));
}

double calkaSek(int n)
{
    double xp, xk, h, calka;
    xp = -1;
    xk = 1;

    h = (xk - xp) / (double)n;

    calka = 0;
    for (int i=1; i<n; i++)
    {
        calka += f(xp + i * h);
    }
    calka += f(xp) / 2;
    calka += f(xk) / 2;
    calka *= h;
    calka *= 2;

    std::cout << "Wynik calkowania sek: " << calka << std::endl;

    return calka;
}

double calkaRow(int n)
{
    int n_w = 100;
    double xp, xk, h, calka;
    xp = -1;
    xk = 1;

    h = (xk - xp) / (double)n;

    double* calk_h = (double*) malloc(n * sizeof(double));
    for(int i = 0; i < n; i++)
    {
        calk_h[i] = 0;
    }
    double* calk_d;
    hipError_t rc = hipMalloc(&calk_d, n * sizeof(double));
    hipMemcpy(calk_d, calk_h, n * sizeof(double), hipMemcpyHostToDevice);
    if (rc != hipSuccess)
        std::cout << hipGetErrorString(rc) << std::endl;
    calkaRowKernel<<<n_w, 1>>>(calk_d, n);


    hipMemcpy(calk_h, calk_d, n * sizeof(double), hipMemcpyDeviceToHost);
    calka = 0;

    for (size_t i = 0; i < n; i++)
    {
        calka += calk_h[i];
    }

    calka += f(xp) / 2;
    calka += f(xk) / 2;
    calka *= h;
    calka *= 2;

    std::cout << "Wynik calkowania row: " << calka << std::endl;
    return calka;
}

int main(int argc, char** argv) {
    
    int n = 1000000;

    calkaSek(n);
    calkaRow(n);

    return 0;
}
